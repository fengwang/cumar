
#include <hip/hip_runtime.h>

__device__ __forceinline__ double dr_xhvpqsohikrkyd( double a, double b ){ return a>b?a:b; }


extern "C" __global__  __launch_bounds__ (128) void gr_xhvpqsohikrkyd(const double * __restrict__ input, double * __restrict__ output)
{
    extern __shared__ double shared_cache[];

    unsigned long const thread_index_in_current_block = threadIdx.x;
    unsigned long const thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long const start_index = thread_index;

    double current_thread_reduction = input[start_index];//thread and block configuration guarantees boundary condition here

    if ( start_index < 140 - 128 )
        current_thread_reduction = dr_xhvpqsohikrkyd( current_thread_reduction, input[start_index+128] );


    shared_cache[thread_index_in_current_block] = current_thread_reduction;

    __syncthreads();

    if (128 > 1024)
    {
        if ( (thread_index_in_current_block < 1024) && (thread_index_in_current_block+1024 < 128) )
            shared_cache[thread_index_in_current_block] = dr_xhvpqsohikrkyd( shared_cache[thread_index_in_current_block], shared_cache[thread_index_in_current_block+1024] );
        __syncthreads();
    }

    if (128 > 512)
    {
        if ( (thread_index_in_current_block < 512) && (thread_index_in_current_block+512 < 128) )
            shared_cache[thread_index_in_current_block] = dr_xhvpqsohikrkyd( shared_cache[thread_index_in_current_block], shared_cache[thread_index_in_current_block+512] );
        __syncthreads();
    }

    if (128 > 256)
    {
        if ( (thread_index_in_current_block < 256) && (thread_index_in_current_block+256 < 128) )
            shared_cache[thread_index_in_current_block] = dr_xhvpqsohikrkyd( shared_cache[thread_index_in_current_block], shared_cache[thread_index_in_current_block+256] );
        __syncthreads();
    }

    if (128 > 128)
    {
        if ( (thread_index_in_current_block < 128) && (thread_index_in_current_block+128 < 128) )
            shared_cache[thread_index_in_current_block] = dr_xhvpqsohikrkyd( shared_cache[thread_index_in_current_block], shared_cache[thread_index_in_current_block+128] );
        __syncthreads();
    }

    if (128 > 64)
    {
        if ( (thread_index_in_current_block < 64) && (thread_index_in_current_block+64 < 128) )
            shared_cache[thread_index_in_current_block] = dr_xhvpqsohikrkyd( shared_cache[thread_index_in_current_block], shared_cache[thread_index_in_current_block+64] );
        __syncthreads();
    }

    if (128 > 32)
    {
        if ( (thread_index_in_current_block < 32) && (thread_index_in_current_block+32 < 128) )
            shared_cache[thread_index_in_current_block] = dr_xhvpqsohikrkyd( shared_cache[thread_index_in_current_block], shared_cache[thread_index_in_current_block+32] );
        __syncthreads();
    }

    if ( (128 > 16) && (thread_index_in_current_block < 16) && (thread_index_in_current_block+16 < 128) )
            shared_cache[thread_index_in_current_block] = dr_xhvpqsohikrkyd( shared_cache[thread_index_in_current_block], shared_cache[thread_index_in_current_block+16] );
    __syncthreads();

    if ( (128 > 8) && (thread_index_in_current_block < 8) && (thread_index_in_current_block+8 < 128) )
            shared_cache[thread_index_in_current_block] = dr_xhvpqsohikrkyd( shared_cache[thread_index_in_current_block], shared_cache[thread_index_in_current_block+8] );
    __syncthreads();

    if ( (128 > 4) && (thread_index_in_current_block < 4) && (thread_index_in_current_block+4 < 128) )
            shared_cache[thread_index_in_current_block] = dr_xhvpqsohikrkyd( shared_cache[thread_index_in_current_block], shared_cache[thread_index_in_current_block+4] );
    __syncthreads();

    if ( (128 > 2) && (thread_index_in_current_block < 2) && (thread_index_in_current_block+2 < 128) )
            shared_cache[thread_index_in_current_block] = dr_xhvpqsohikrkyd( shared_cache[thread_index_in_current_block], shared_cache[thread_index_in_current_block+2] );
    __syncthreads();

    if ( (128 > 1) && (thread_index_in_current_block < 1) && (thread_index_in_current_block+1 < 128) )
            shared_cache[thread_index_in_current_block] = dr_xhvpqsohikrkyd( shared_cache[thread_index_in_current_block], shared_cache[thread_index_in_current_block+1] );
    __syncthreads();

    if (thread_index_in_current_block == 0) output[blockIdx.x] = shared_cache[0];
}
